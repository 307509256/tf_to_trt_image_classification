#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 * Full license terms provided in LICENSE.md file.
 */
#include <string>
#include <cstring>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <NvInfer.h>
#include <opencv2/opencv.hpp>
#include "examples/classify_image/utils.h"

using namespace std;
using namespace cv;
using namespace nvinfer1;
int telephone = 0;
class Logger : public ILogger
{
    void log(Severity severity, const char *msg) override
    {
        if (severity != Severity::kINFO)
            cout << msg << endl;
    }
} gLogger;

/**
 * image_file: path to image
 * plan_file: path of the serialized engine file
 * label_file: file with <class_name> per line
 * input_name: name of the input tensor
 * output_name: name of the output tensor
 * preprocessing_fn: 'vgg' or 'inception'
 */
int main(int argc, char *argv[])
{
    if (argc != 6)
    {
        cout << "Usage: classify_image <image_file> <plan_file> <label_file> <input_name> <output_name> <preprocessing_fn>\n";
        return 0;
    }

    //string videoFilename = argv[1];
    string planFilename = argv[1];
    string labelFilename = argv[2];
    string inputName = argv[3];
    string outputName = argv[4];
    string preprocessingFn = argv[5];

    /* load the engine */
    cout << "Loading TensorRT engine from plan file..." << endl;
    ifstream planFile(planFilename);

    if (!planFile.is_open())
    {
        cout << "Could not open plan file." << endl;
        return 1;
    }

    stringstream planBuffer;
    planBuffer << planFile.rdbuf();
    string plan = planBuffer.str();
    IRuntime *runtime = createInferRuntime(gLogger);
    ICudaEngine *engine = runtime->deserializeCudaEngine((void *)plan.data(), plan.size(), nullptr);
    IExecutionContext *context = engine->createExecutionContext();

    /* get the input / output dimensions */
    int inputBindingIndex, outputBindingIndex;
    inputBindingIndex = engine->getBindingIndex(inputName.c_str());
    outputBindingIndex = engine->getBindingIndex(outputName.c_str());

    if (inputBindingIndex < 0)
    {
        cout << "Invalid input name." << endl;
        return 1;
    }

    if (outputBindingIndex < 0)
    {
        cout << "Invalid output name." << endl;
        return 1;
    }

    Dims inputDims, outputDims;
    inputDims = engine->getBindingDimensions(inputBindingIndex);
    outputDims = engine->getBindingDimensions(outputBindingIndex);
    int inputWidth, inputHeight;
    inputHeight = inputDims.d[1];
    inputWidth = inputDims.d[2];

    /* read image, convert color, and resize */
    cout << "Preprocessing Video input..." << endl;
    VideoCapture cap;
    cap.open(1);
    cap.set(CV_CAP_PROP_FPS, 5);
    //VideoCapture cap(0);

    // board
    //VideoCapture cap("nvcamerasrc ! 'video/x-raw(memory:NVMM), width=(int)1280, height=(int)720,format=(string)NV12' ! nvvidconv flip-method=0 ! video/x-raw ! appsink name=mysink");
    //VideoCapture cap("nvcamerasrc ! 'video/x-raw(memory:NVMM), width=640, height=480, framerate=30/1, format=NV12' ! nvvidconv flip-method=0 ! nvegltransform ! nveglglessink -e");
    //usb camera
    //VideoCapture cap("v4l2src device=/dev/video1 ! video/x-raw, width=(int)1280, height=(int)720, format=RGB ! videoconvert ! video/x-raw, format=RGB ! videoconvert ! appsink name=mysink");

    //check if camera opened successfully
    if (!cap.isOpened())
    {
        cout << "Error opening video stream or file" << endl;
        return -1;
    }
    for (;;)
    {

        cv::Mat image;

        cap >> image;

        if (image.empty())
        {
            cout << "Could not read image from file." << endl;
            return 1;
        }
        cv::imshow("Frame", image);
        cv::cvtColor(image, image, cv::COLOR_BGR2RGB, 3);
        cv::resize(image, image, cv::Size(inputWidth, inputHeight));

        /* convert from uint8+NHWC to float+NCHW */
        float *inputDataHost, *outputDataHost;
        size_t numInput, numOutput;
        numInput = numTensorElements(inputDims);
        numOutput = numTensorElements(outputDims);
        inputDataHost = (float *)malloc(numInput * sizeof(float));
        outputDataHost = (float *)malloc(numOutput * sizeof(float));
        cvImageToTensor(image, inputDataHost, inputDims);
        if (preprocessingFn == "vgg")
            preprocessVgg(inputDataHost, inputDims);
        else if (preprocessingFn == "inception")
            preprocessInception(inputDataHost, inputDims);
        else
        {
            cout << "Invalid preprocessing function argument, must be vgg or inception. \n"
                 << endl;
            return 1;
        }

        /* transfer to device */
        float *inputDataDevice, *outputDataDevice;
        hipMalloc(&inputDataDevice, numInput * sizeof(float));
        hipMalloc(&outputDataDevice, numOutput * sizeof(float));
        hipMemcpy(inputDataDevice, inputDataHost, numInput * sizeof(float), hipMemcpyHostToDevice);
        void *bindings[2];
        bindings[inputBindingIndex] = (void *)inputDataDevice;
        bindings[outputBindingIndex] = (void *)outputDataDevice;

        /* execute engine */
        //cout << "Executing inference engine..." << endl;
        const int kBatchSize = 1;
        context->execute(kBatchSize, bindings);

        /* transfer output back to host */
        hipMemcpy(outputDataHost, outputDataDevice, numOutput * sizeof(float), hipMemcpyDeviceToHost);

        /* parse output */
        vector<size_t> sortedIndices = argsort(outputDataHost, outputDims);

        /* cout << "\nThe top-5 indices are: ";
        for (int i = 0; i < 5; i++)
            cout << sortedIndices[i] << " ";
        */
        ifstream labelsFile(labelFilename);

        if (!labelsFile.is_open())
        {
            cout << "\nCould not open label file." << endl;
            return 1;
        }

        vector<string> labelMap;
        string label;
        while (getline(labelsFile, label))
        {
            labelMap.push_back(label);
        }

        //cout << "\nWhich corresponds to class labels: ";
        for (int i = 0; i < 5; i++)
	{
            string str = labelMap[sortedIndices[i]];
            size_t fi= str.find("telephone", 0);
            if (fi != std::string::npos)
            {
            	cout << endl
                 << i << ". " << "telephone, num = " << telephone << ",score = " << outputDataHost[sortedIndices[i]] << ",label = " << sortedIndices[i];  //label is begin 0, phone_id is 488;
            	cout << endl;
                telephone ++;
	    }
	}
        /* clean up */
        //runtime->destroy();
        //engine->destroy();
        //context->destroy();
        free(inputDataHost);
        free(outputDataHost);
        hipFree(inputDataDevice);
        hipFree(outputDataDevice);
        char c = (char)waitKey(25);
        if (c == 27)
            break;
    }
    cap.release();
    // Closes all the frames

    return 0;
}
